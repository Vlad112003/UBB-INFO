#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <chrono>

using namespace std;

#define CHECK_CUDA(call)                                                    \
    {                                                                       \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            cerr << "CUDA error in " << __FILE__ << ":" << __LINE__        \
                 << " - " << hipGetErrorString(err) << endl;              \
            exit(err);                                                      \
        }                                                                   \
    }

const int BLOCK_SIZE = 16;

int N, M, p, compare;
int n = 3;
vector<vector<int>> mat, convMat;

void readInput() {
    ifstream inputFile("date.txt");
    if (!inputFile) {
        cerr << "Error opening file." << endl;
        return;
    }
    mat.resize(N, vector<int>(M));
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            inputFile >> mat[i][j];
        }
    }
    // 1 0 1
    // 0 2 0
    // -1 0 -1
    convMat.resize(n, vector<int>(n));
    convMat.assign({{1, 0, 1}, {0, 2, 0}, {-1, 0, -1}});

    inputFile.close();
}

void writeOutput() {
    ofstream outputFile("output.txt");
    if (!outputFile) {
        cerr << "Error opening file." << endl;
        return;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            outputFile << mat[i][j];
            if (j < M - 1) {
                outputFile << " ";
            }
        }
        if (i < N - 1) {
            outputFile << endl;
        }
    }
    outputFile.close();
}

// CUDA kernel for convolution
__global__ void convolveKernel(int *d_mat, int *d_convMat, int *d_result, int N, int M, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) {
                int x = row - n / 2 + i;
                int y = col - n / 2 + j;
                if (x >= 0 && y >= 0 && x < N && y < M) {
                    sum += d_mat[x * M + y] * d_convMat[i * n + j];
                }
            }
        }
        d_result[row * M + col] = sum;
    }
}

void linii() {
    size_t matSize = N * M * sizeof(int);
    size_t convMatSize = n * n * sizeof(int);

    // Allocate device memory
    int *d_mat, *d_convMat, *d_result;
    CHECK_CUDA(hipMalloc(&d_mat, matSize));
    CHECK_CUDA(hipMalloc(&d_convMat, convMatSize));
    CHECK_CUDA(hipMalloc(&d_result, matSize));

    // Flatten host matrices for device transfer
    vector<int> flatMat(N * M);
    vector<int> flatConvMat(n * n);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            flatMat[i * M + j] = mat[i][j];
        }
    }
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            flatConvMat[i * n + j] = convMat[i][j];
        }
    }

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_mat, flatMat.data(), matSize, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_convMat, flatConvMat.data(), convMatSize, hipMemcpyHostToDevice));

    // Define grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((M + 15) / 16, (N + 15) / 16);

    // Launch the kernel
    convolveKernel<<<gridSize, blockSize>>>(d_mat, d_convMat, d_result, N, M, n);
    CHECK_CUDA(hipGetLastError());

    // Copy results back to host
    vector<int> flatResult(N * M);
    CHECK_CUDA(hipMemcpy(flatResult.data(), d_result, matSize, hipMemcpyDeviceToHost));

    // Reshape flatResult into 2D result matrix
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            mat[i][j] = flatResult[i * M + j];
        }
    }

    // Free device memory
    CHECK_CUDA(hipFree(d_mat));
    CHECK_CUDA(hipFree(d_convMat));
    CHECK_CUDA(hipFree(d_result));
}

void secvential() {

    vector<int> auxRow(M + n - 1);

    // calculate new values for border cells and save them in separate vectors
    for (int i = 0; i < N; ++i) {
        for (int j = -n / 2; j < M + n / 2; ++j) {
            int x = max(0, min(i, N - 1));
            int y = max(0, min(j, M - 1));
            auxRow[j + n / 2] = mat[x][y];
        }
        // calculate new values for non-border cells and save them directly in the matrix
        for (int j = 0; j < M; ++j) {
            int sum = 0;
            for (int i1 = 0; i1 < n; ++i1) {
                for (int j1 = 0; j1 < n; ++j1) {
                    int x = i - n / 2 + i1;
                    int y = j - n / 2 + j1;
                    if (x >= 0 && y >= 0 && x < N && y < M) {
                        sum += auxRow[y + n / 2] * convMat[i1][j1];
                    }
                }
            }
            mat[i][j] = sum;
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        cerr << "Usage: " << argv[0] << " <matrix_rows> <matrix_cols> <function_name: secvential/linii> <compare-1/0>" << endl;
        return 1;
    }

    // p = stoi(argv[1]);
    N = stoi(argv[1]);
    M = stoi(argv[2]);
    string functionName = argv[3];
    n = 3;
    compare = stoi(argv[4]);

    // block folosit pentru testare
    // p = 8;
    // N = 100;
    // M = 100;
    // string functionName = "linii";
    // n = 3;
    // compare = 0;

    auto start = chrono::high_resolution_clock::now();
    readInput();

    if (functionName == "secvential") {
        secvential();
    } else if (functionName == "linii") {
        linii();
    } else {
        cerr << "Invalid function name. Use one of: secvential, linii" << endl;
        return 1;
    }

    auto stop = chrono::high_resolution_clock::now();
    chrono::duration<double, std::micro> duration = stop - start;
    double microseconds = duration.count();
    double milliseconds = microseconds / 1000;

    cout << milliseconds << " milliseconds" << endl;

    writeOutput();

    if (compare == 1) {
        if (N != 100 && N != 1000 && N != 10000) {
            cout << "Invalid matrix size" << endl;
            return 1;
        }

        ifstream outputFile("output.txt");
        ifstream expectedFile = ifstream("expected" + to_string(N) + "x" + to_string(N) + ".txt");

        string outputLine, expectedLine;
        while (getline(outputFile, outputLine) && getline(expectedFile, expectedLine)) {
            outputLine.erase(outputLine.find_last_not_of(" \n\r\t") + 1);
            expectedLine.erase(expectedLine.find_last_not_of(" \n\r\t") + 1);
            if (outputLine != expectedLine) {
                cout << "Output is not correct" << endl;
                return 1;
            }
        }
        cout << "Output is correct" << endl;
    }

    return 0;
}